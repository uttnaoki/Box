#include "hip/hip_runtime.h"
__global__ void gpuTK(double*fimg,unsigned char*img,int imW,int imH){
#define getpix(x,y) img[((x)+imW*(y))*3+1]
  int x=blockDim.x*blockIdx.x+threadIdx.x;
  int y=blockDim.y*blockIdx.y+threadIdx.y;
  int u,v,W=7;

  if(W+1<=y && y<imH-W-1){
    if(W+1<=x && x<imW-W-1){
      float ix,iy,ixx,ixy,iyy,lamd;
      ixx=iyy=ixy=0;
      for(v=-W;v<=W;v++)
	for(u=-W;u<=W;u++){
	  ix=getpix(x+u+1,y+v)-getpix(x+u-1,y+v);
	  iy=getpix(x+u,y+v+1)-getpix(x+u,y+v-1);
	  ixx+=ix*ix; // <-- incomplete
	  ixy+=ix*iy;
	  iyy+=iy*iy;
	}
	lamd=((ixx+iyy)-sqrt(pow(ixx+iyy,2)-4*(ixx*iyy-pow(ixy,2))))/2;
      fimg[x+imW*y]=lamd; // <-- incomplete
    }
  }else fimg[x+imW*y]=0;
}

#include"image.h"

typedef struct {
  double *data;
  int W,H;
} Matrix;

// TKfilter.c では ImageFeature 本体を除去して，
// プロトタイプ宣言 void ImageFeature(Matrix*im2,Image*im); のみを書く．

extern "C"
void ImageFeature(Matrix*im2,Image*im){
  double*d_dst;
  unsigned char*d_src;
  hipMalloc(&d_src,im->W*im->H*3);
  hipMalloc(&d_dst,sizeof(double)*im->W*im->H);
  hipMemcpy(d_src,im->data,im->W*im->H*3,hipMemcpyHostToDevice);
  gpuTK<<<dim3((im->W+15)/16,(im->H+15)/16),dim3(16,16)>>>(d_dst,d_src,im->W,im->H);
  hipMemcpy(im2->data,d_dst,im->W*im->H*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(d_dst);
  hipFree(d_src);
}