#include "hip/hip_runtime.h"
#include"image.h"
#define getpix(x,y) img[((x)+imW*(y))*3+1]

__global__ void gpuTK_vertical(float*tmp,unsigned char *img,int imW,int imH){
  int x=blockDim.x*blockIdx.x+threadIdx.x;
  int y=blockDim.y*blockIdx.y+threadIdx.y;
  int v,W=7;
  int mat=imW*imH;

  if(W+1<=y && y<imH-W-1)
    if(1<=x && x<imW-1){
      float ix,iy,ixx,ixy,iyy;
      ixx=iyy=ixy=0;
      for(v=-W;v<=W;v++){
	ix=getpix(x+1,y+v)-getpix(x-1,y+v);
	iy=getpix(x,y+v+1)-getpix(x,y+v-1);
	ixx+=ix*ix;
	ixy+=ix*iy;
	iyy+=iy*iy;
      }
      tmp[(x+imW*y)]=ixx;
      tmp[(x+imW*y+mat)]=ixy;
      tmp[(x+imW*y+mat*2)]=iyy;
    }
}

__global__ void gpuTK_horizontal(double*fimg,float*tmp,int imW,int imH){
  int x=blockDim.x*blockIdx.x+threadIdx.x;
  int y=blockDim.y*blockIdx.y+threadIdx.y;
  int u,W=7;
  int mat=imW*imH;

  if(W+1<=y && y<imH-W-1 &&
     W+1<=x && x<imW-W-1){
      float ixx,ixy,iyy;
      double lamd;
      ixx=iyy=ixy=0;
      for(u=-W;u<=W;u++){
	ixx+=tmp[(x+u+imW*y)];
	ixy+=tmp[(x+u+imW*y+mat)];
	iyy+=tmp[(x+u+imW*y+mat*2)];
      }
      lamd=((ixx+iyy)-sqrt(pow(ixx+iyy,2)-4*(ixx*iyy-ixy*ixy)))/2;
      fimg[x+imW*y]=lamd;
    }else fimg[x+imW*y]=0;
}

// TKfilter.c では ImageFeature 本体を除去して，
// プロトタイプ宣言 void ImageFeature(Matrix*im2,Image*im); のみを書く．

extern "C"
void ImageFeature(Matrix*im2,Image*im){
  double*d_dst;
  float *d_tmp;
  unsigned char*d_src;
  hipMalloc(&d_src,im->W*im->H*3);
  hipMalloc(&d_dst,sizeof(double)*im->W*im->H);
  hipMalloc(&d_tmp,sizeof(float)*im->W*im->H*3);
  hipMemcpy(d_src,im->data,im->W*im->H*3,hipMemcpyHostToDevice);
  gpuTK_vertical<<<dim3((im->W+15)/16,(im->H+15)/16),dim3(16,16)>>>(d_tmp,d_src,im->W,im->H);
  gpuTK_horizontal<<<dim3((im->W+15)/16,(im->H+15)/16),dim3(16,16)>>>(d_dst,d_tmp,im->W,im->H);  
  hipMemcpy(im2->data,d_dst,im->W*im->H*sizeof(double),hipMemcpyDeviceToHost);
  hipFree(d_dst);
  hipFree(d_src);
  hipFree(d_tmp);
}